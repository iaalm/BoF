#include<stdio.h>
#include<stdlib.h>
//#include<string.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define INF (64 * 64 * 128 * 2)
#define N_FEATURE (128)
typedef float fv[N_FEATURE];

static void HandleError( hipError_t err,
        const char *file,
        int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ static void kmeans_cluster(int max_iter, fv *points,fv *center, int n_line, int n_clusters){
    int i,j,k,l,q;
    extern __shared__ unsigned int cluster[];
    __shared__ int p;
    float m,n;
    fv tfeature;
    for(l = 0; l < max_iter;l++){
        if(threadIdx.x == 0)
            p = 0;
        __syncthreads();
        for(i = threadIdx.x;i < n_line;i+=blockDim.x){
            //memcpy(tfeature,points[i],sizeof(float)*N_FEATURE);
            for(j = 0; j < N_FEATURE;j++)
                tfeature[j] = points[i][j];
            n = INF;   
            for(j = 0; j < n_clusters;j++){
                m = 0;
                for(k = 0;k < N_FEATURE;k++)
                    m += (tfeature[k] - center[j][k]) * (tfeature[k] - center[j][k]);
                if(m < n){
                    q = j;
                    n = m;
                }
            }
            //sum += n;
            if(q != cluster[i]){
                cluster[i] = q;
                if(p == 0)
                    atomicAdd(&p, 1);

            }
        }
        __syncthreads();
        if(p == 0){
            return ;
        }
        for(i = threadIdx.x;i < n_clusters;i+=blockDim.x){
            for(j = 0; j < N_FEATURE;j++)
                tfeature[j] = 0;
            n = 0;

            for (j = 0; j < n_line;j++)
                if(cluster[j] == i){
                    n++;
                    for(k = 0;k < N_FEATURE;k++)
                        tfeature[k] += points[j][k];
                }
            if(n > 0)
                for(j = 0; j < N_FEATURE;j++)
                    tfeature[j] /= n;
            //really need else to add point
            tfeature[0] = n;
            //memcpy(center[i],tfeature,sizeof(float)*N_FEATURE);
            for(j = 0; j < N_FEATURE;j++)
                center[i][j] = tfeature[j];
        }
    }
}

int main(int argc, char* argv[]){
    float sum;
    int n_clusters = atoi(argv[1]);
    fv* points;
    fv center[n_clusters];
    int n_line = 0, i, j, k, l, p, q;
    int tab[n_clusters];
    float m, n;
    char str[256];
    int b_continue = 0;

    if(argc > 4 && strcmp(argv[4],"-c") == 0)
        b_continue = 1;

    FILE *fp_list = fopen(argv[2], "r"), *fp_file;
    fgets(str, 256, fp_list);
    while(!feof(fp_list)){
        str[strlen(str) - 1] = '\0';
        fp_file = fopen(str, "r");
        fscanf(fp_file, "%*d%d", &j);
        n_line += j;
        fclose(fp_file);
        fgets(str, 256, fp_list);
    }
    points =  (fv*)malloc(sizeof(float) * N_FEATURE * n_line);
    printf("total %d lines\n",n_line);

    l = 0;
    rewind(fp_list);
    fgets(str, 256, fp_list);
    while(!feof(fp_list)){
        str[strlen(str) - 1] = '\0';
        fp_file = fopen(str, "r");
        fscanf(fp_file, "%*d%d", &j);
        for(i = 0;i < j;i++){
            fscanf(fp_file, "%*f%*f%*f%*f%*f");
            q = 0;
            for(k = 0;k < N_FEATURE;k++){
                fscanf(fp_file, "%d", &p);
                q += p;
                points[l + i][k] = sqrtf(p); // sqrt it
            }
            m = sqrtf(q);
            if(q > 0)
                for(k = 0;k < N_FEATURE;k++){
                    points[l + i][k] /= m;        //l2 normal
                }
        }
        l += j;
        fclose(fp_file);
        fgets(str, 256, fp_list);
    }
    fclose(fp_list);

    puts("data load done!");
    //init center
    if(b_continue){
        fp_list = fopen(argv[3], "r");
        for(i = 0;i < n_clusters;i++){
            for(j = 0;j < N_FEATURE;j++)
                fscanf(fp_list, "%f",&center[i][j]);
        }
        fclose(fp_list);
    }
    else{
        for(i = 0;i < n_clusters;i++)
            for(j = 0;j < N_FEATURE;j++)
                center[i][j] = random() / (float) RAND_MAX;
    }

    //kmeans
    fv* dev_points;
    fv* dev_center;
    HANDLE_ERROR( hipMalloc( (void**)&dev_center, n_clusters * N_FEATURE * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_points, n_line * N_FEATURE * sizeof(float) ) );
    HANDLE_ERROR( hipMemcpy( dev_center, center, n_clusters * N_FEATURE * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_points, points, n_line * N_FEATURE * sizeof(float), hipMemcpyHostToDevice ) );
    puts("running cuda.");
    kmeans_cluster<<<1,2,n_line * sizeof(float)>>>(1000,dev_points,dev_center,n_line,n_clusters);
    HANDLE_ERROR( hipMemcpy( center, dev_center, n_clusters * N_FEATURE * sizeof(float), hipMemcpyDeviceToHost ) );

    fp_list = fopen(argv[3], "w");
    for(i = 0;i < n_clusters;i++){
        for(j = 0;j < N_FEATURE;j++)
            fprintf(fp_list, "%f ",center[i][j]);
        fprintf(fp_list,"\n");
    }
    fclose(fp_list);
    free(points);
    HANDLE_ERROR( hipFree( dev_points ) );
    HANDLE_ERROR( hipFree( dev_center ) );

    return p;
}

