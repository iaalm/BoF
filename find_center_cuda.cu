#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define INF (64 * 64 * 128 * 2)
#define N_FEATURE (128)

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct point{
    float f[N_FEATURE];
    int c;
};

__global__ static void kmeans_cluster(point* points,float (*center)[N_FEATURE], int n_line, int n_clusters, int* p){
    int i,j,k,q;
    float m,n;
        for(i = blockIdx.x;i < n_line;i+=gridDim.x){
            n = INF;   
            for(j = 0; j < n_clusters;j++){
                m = 0;
                for(k = 0;k < N_FEATURE;k++)
                    m += (points[i].f[k] - center[j][k]) * (points[i].f[k] - center[j][k]);
                if(m < n){
                    q = j;
                    n = m;
                }
            }
            //sum += n;
            if(q != points[i].c && *p == 0){
                #pragma omp crititcal
                {
                    *p = 1;
                }
                points[i].c = q;
            }
        }
}
__global__ static void kmeans_center(point* points,float (*center)[N_FEATURE], int n_line, int n_clusters){
    int i,j,k,n;
        for(i = blockIdx.x;i < n_clusters;i+=gridDim.x){
            for(j = 0; j < N_FEATURE;j++)
                center[i][j] = 0;
            n = 0;

            for (j = 0; j < n_line;j++)
                if(points[j].c == i){
                    n++;
                    for(k = 0;k < N_FEATURE;k++)
                        center[i][k] += points[i].f[k];
                }
            for(j = 0; j < N_FEATURE;j++)
                center[i][j] /= n;
        }
}

int main(int argc, char* argv[]){
    float sum;
    int n_clusters = atoi(argv[1]);
    point* points;
    float center[n_clusters][N_FEATURE];
    int n_line = 0, i, j, k, l, p, q;
    int tab[n_clusters];
    float m, n;
    char str[256];
    int b_continue = 0;

    if(argc > 4 && strcmp(argv[4],"-c") == 0)
        b_continue = 1;

    FILE *fp_list = fopen(argv[2], "r"), *fp_file;
    fgets(str, 256, fp_list);
    while(!feof(fp_list)){
        str[strlen(str) - 1] = '\0';
        fp_file = fopen(str, "r");
        fscanf(fp_file, "%*d%d", &j);
        n_line += j;
        fclose(fp_file);
        fgets(str, 256, fp_list);
    }
    points = (point*) malloc(sizeof(point) * n_line);
    printf("total %d lines\n",n_line);

    l = 0;
    rewind(fp_list);
    fgets(str, 256, fp_list);
    while(!feof(fp_list)){
        str[strlen(str) - 1] = '\0';
        fp_file = fopen(str, "r");
        fscanf(fp_file, "%*d%d", &j);
        for(i = 0;i < j;i++){
            fscanf(fp_file, "%*f%*f%*f%*f%*f");
            q = 0;
            for(k = 0;k < N_FEATURE;k++){
                fscanf(fp_file, "%d", &p);
                q += p;
                points[l + i].f[k] = sqrtf(p); // sqrt it
            }
            m = sqrtf(q);
            if(q > 0)
                for(k = 0;k < N_FEATURE;k++){
                    points[l + i].f[k] /= m;        //l2 normal
                }
        }
        l += j;
        fclose(fp_file);
        fgets(str, 256, fp_list);
    }
    fclose(fp_list);

    //init center
    if(b_continue){
        fp_list = fopen(argv[3], "r");
        for(i = 0;i < n_clusters;i++){
            for(j = 0;j < N_FEATURE;j++)
                fscanf(fp_list, "%f",&center[i][j]);
        }
        fclose(fp_list);
    }
    else{
        for(i = 0;i < n_clusters;i++)
            for(j = 0;j < N_FEATURE;j++)
                center[i][j] = random() / (float) RAND_MAX;
    }

    //kmeans
    point* dev_points;
    float (*dev_center)[N_FEATURE];
    int *dev_p;
    HANDLE_ERROR( hipMalloc( (void**)&dev_center, n_clusters * N_FEATURE * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_points, n_line * sizeof(point) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_p, sizeof(int) ) );
    HANDLE_ERROR( hipMemcpy( dev_points, points, n_line * sizeof(point), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_center, center, n_clusters * N_FEATURE * sizeof(float), hipMemcpyHostToDevice ) );
    for(l = 0; l < 1000;l++){
        // start
        // sum = 0
        p = 0;
    HANDLE_ERROR( hipMemcpy( dev_p, &p, sizeof(int), hipMemcpyHostToDevice ) );
    kmeans_cluster<<<128,1>>>(dev_points,dev_center,n_line,n_clusters,dev_p);
    HANDLE_ERROR( hipMemcpy( &p, dev_p, sizeof(int), hipMemcpyDeviceToHost ) );
        //printf("loop %6d: %f\n",l,sum/n_line);

        if(!p)
            break;
        //calc center
    kmeans_center<<<128,1>>>(dev_points,dev_center,n_line,n_clusters);
    printf("loop %4d: %f\n",l,sum/n_line);
    }
    HANDLE_ERROR( hipMemcpy( center, dev_center, n_clusters * N_FEATURE * sizeof(float), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipFree( dev_points ) );
    HANDLE_ERROR( hipFree( dev_center ) );
    HANDLE_ERROR( hipFree( dev_p ) );

    fp_list = fopen(argv[3], "w");
    for(i = 0;i < n_clusters;i++){
        for(j = 0;j < N_FEATURE;j++)
            fprintf(fp_list, "%f ",center[i][j]);
        fprintf(fp_list,"\n");
    }
    fclose(fp_list);
    delete points;

    return p;
}

